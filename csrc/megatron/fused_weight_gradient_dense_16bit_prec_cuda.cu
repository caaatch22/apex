#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "type_shim.h"

/* Includes, HIP */
#include <hipblaslt/hipblaslt.h>
#include <hipblaslt/hipblaslt-ext.hpp>

#ifndef CHECK_HIP_ERROR
#define CHECK_HIP_ERROR(error)                    \
    if(error != hipSuccess)                       \
    {                                             \
        fprintf(stderr,                           \
                "Hip error: '%s'(%d) at %s:%d\n", \
                hipGetErrorString(error),         \
                error,                            \
                __FILE__,                         \
                __LINE__);                        \
        exit(EXIT_FAILURE);                       \
    }
#endif
#ifndef CHECK_HIPBLASLT_ERROR
#define CHECK_HIPBLASLT_ERROR(error)                                                      \
    if(error != HIPBLAS_STATUS_SUCCESS)                                                   \
    {                                                                                     \
        fprintf(stderr, "hipBLASLt error(Err=%d) at %s:%d\n", error, __FILE__, __LINE__); \
        fprintf(stderr, "\n");                                                            \
        exit(EXIT_FAILURE);                                                               \
    }
#endif

// BF16 inputs and BF16 accumulation
void gemmex_wrapper_fp16(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int batch_count,
    float& alpha,
    float& beta,
    at::BFloat16* A,
    at::BFloat16* B,
    at::BFloat16* C,
    at::BFloat16* D,
    void*   d_workspace,
    int64_t  max_workspace_size,
    hipStream_t   stream) 
{
    hipblasLtMatrixLayout_t matA, matB, matC, matD;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matA, HIP_R_16BF, m, k, m));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matB, HIP_R_16BF, n, k, n));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matC, HIP_R_16BF, m, n, m));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matD, HIP_R_16BF, m, n, m));

    hipblasLtMatmulDesc_t matmul;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescCreate(&matmul, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    // Set User Preference attributes
    hipblasLtMatmulPreference_t pref;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulPreferenceCreate(&pref));
    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulPreferenceSetAttribute(pref,
                                              HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                              &max_workspace_size,
                                              sizeof(max_workspace_size)));

    const int                        request_solutions = 1;
    hipblasLtMatmulHeuristicResult_t heuristicResult[request_solutions];
    int                              returnedAlgoCount = 0;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle,
                                                          matmul,
                                                          matA,
                                                          matB,
                                                          matC,
                                                          matD,
                                                          pref,
                                                          request_solutions,
                                                          heuristicResult,
                                                          &returnedAlgoCount));

    if(returnedAlgoCount == 0)
    {
        std::cerr << "No valid solution found!" << std::endl;
        return;
    }

    uint64_t workspace_size = 0;
    for(int i = 0; i < returnedAlgoCount; i++)
        workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);

    CHECK_HIPBLASLT_ERROR(hipblasLtMatmul(handle,
                                          matmul,
                                          &alpha,
                                          A,
                                          matA,
                                          B,
                                          matB,
                                          &beta,
                                          C,
                                          matC,
                                          D,
                                          matD,
                                          &heuristicResult[0].algo,
                                          d_workspace,
                                          workspace_size,
                                          stream));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matA));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matB));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matC));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matD));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulPreferenceDestroy(pref));
    return;
}

// FP16 inputs and FP16 accumulation
void gemmex_wrapper_fp16(
    hipblasLtHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int batch_count,
    float& alpha,
    float& beta,
    at::Half* A,
    at::Half* B,
    at::Half* C,
    at::Half* D,
    void*   d_workspace,
    int64_t  max_workspace_size,
    hipStream_t   stream) 
{
    hipblasLtMatrixLayout_t matA, matB, matC, matD;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matA, HIP_R_16F, m, k, m));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matB, HIP_R_16F, n, k, n));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matC, HIP_R_16F, m, n, m));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matD, HIP_R_16F, m, n, m));

    hipblasLtMatmulDesc_t matmul;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescCreate(&matmul, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(
        matmul, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    // Set User Preference attributes
    hipblasLtMatmulPreference_t pref;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulPreferenceCreate(&pref));
    CHECK_HIPBLASLT_ERROR(
        hipblasLtMatmulPreferenceSetAttribute(pref,
                                              HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                              &max_workspace_size,
                                              sizeof(max_workspace_size)));

    const int                        request_solutions = 1;
    hipblasLtMatmulHeuristicResult_t heuristicResult[request_solutions];
    int                              returnedAlgoCount = 0;
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle,
                                                          matmul,
                                                          matA,
                                                          matB,
                                                          matC,
                                                          matD,
                                                          pref,
                                                          request_solutions,
                                                          heuristicResult,
                                                          &returnedAlgoCount));

    if(returnedAlgoCount == 0)
    {
        std::cerr << "No valid solution found!" << std::endl;
        return;
    }

    uint64_t workspace_size = 0;
    for(int i = 0; i < returnedAlgoCount; i++)
        workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);

    CHECK_HIPBLASLT_ERROR(hipblasLtMatmul(handle,
                                          matmul,
                                          &alpha,
                                          A,
                                          matA,
                                          B,
                                          matB,
                                          &beta,
                                          C,
                                          matC,
                                          D,
                                          matD,
                                          &heuristicResult[0].algo,
                                          d_workspace,
                                          workspace_size,
                                          stream));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matA));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matB));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matC));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matD));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul));
    CHECK_HIPBLASLT_ERROR(hipblasLtMatmulPreferenceDestroy(pref));
    return;
}

template <typename T>
void wgrad_gemm_accum_fp16_cuda(T *input, T *d_output, T *d_weight,int in_dim, int hidden_dim, int out_dim) {

    hipblasLtHandle_t handle = at::cuda::getCurrentCUDABlasLtHandle();
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    float alpha = 1.0;
    float beta  = 1.0;
    const int batch_count = 1;
    void*   d_workspace;
    int64_t max_workspace_size = 32*1024*1024;
    if(max_workspace_size > 0)
        CHECK_HIP_ERROR(hipMalloc(&d_workspace, max_workspace_size));
    gemmex_wrapper_fp16(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,        //m
        out_dim,       //n
        hidden_dim,    //k
        batch_count,
        alpha,
        beta,
        input,         //da   
        d_output,      //db
        d_weight,      //dc
        d_weight,      //dd
        d_workspace,
        max_workspace_size,
        stream);

} 

template void wgrad_gemm_accum_fp16_cuda<at::Half>(at::Half *input, at::Half *d_output, at::Half *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp16_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, at::BFloat16 *d_weight,  int in_dim, int hidden_dim, int out_dim);

void wgrad_gemm_accum_fp16_cuda_stub(
  at::Tensor &input,
  at::Tensor &d_output,
  at::Tensor &d_weight
) {
    at::Tensor input_2d, d_output_2d;
    // input tensor: collapse to the first dim
    auto in_sizes = input.sizes();
    if (input.dim() > 2) {
        input_2d = input.view({-1, in_sizes[in_sizes.size() - 1]});
    } else {
        input_2d = input;
    }
    // d_output tensor: collapse to the first dim
    auto d_out_sizes = d_output.sizes();
    if (d_output.dim() > 2) {
        d_output_2d = d_output.view({-1, d_out_sizes[d_out_sizes.size() - 1]});
    } else {
        d_output_2d = d_output;
    }

    const int hidden_dim = input_2d.size(0);  //k
    const int in_dim = input_2d.size(1);      //m
    const int out_dim = d_weight.size(0);     //n

    DISPATCH_HALF_AND_BFLOAT(input_2d.scalar_type(), "wgrad_gemm_accum_fp16",
        wgrad_gemm_accum_fp16_cuda<scalar_t>(
            input_2d.data_ptr<scalar_t>(),
            d_output_2d.data_ptr<scalar_t>(),
            d_weight.data_ptr<scalar_t>(), 
            in_dim,
            hidden_dim,
            out_dim);
    );
}
