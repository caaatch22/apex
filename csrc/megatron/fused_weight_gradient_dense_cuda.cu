#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "type_shim.h"

/* Includes, HIP */
#include <hipblaslt/hipblaslt-ext.hpp>

#ifndef CHECK_HIP_ERROR
#define CHECK_HIP_ERROR(error)                    \
    if(error != hipSuccess)                       \
    {                                             \
        fprintf(stderr,                           \
                "Hip error: '%s'(%d) at %s:%d\n", \
                hipGetErrorString(error),         \
                error,                            \
                __FILE__,                         \
                __LINE__);                        \
        exit(EXIT_FAILURE);                       \
    }
#endif

#ifndef CHECK_HIPBLASLT_ERROR
#define CHECK_HIPBLASLT_ERROR(error)                                                      \
    if(error != HIPBLAS_STATUS_SUCCESS)                                                   \
    {                                                                                     \
        fprintf(stderr, "hipBLASLt error(Err=%d) at %s:%d\n", error, __FILE__, __LINE__); \
        fprintf(stderr, "\n");                                                            \
        exit(EXIT_FAILURE);                                                               \
    }
#endif

// BF16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasLtHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int batch_count,
    float& alpha,
    float& beta,
    at::BFloat16* A,
    at::BFloat16* B,
    float*    C,
    float*    D,
    void*     d_workspace,
    int64_t   max_workspace_size,
    hipStream_t   stream) {

    hipblaslt_ext::GemmPreference gemmPref;
    gemmPref.setMaxWorkspaceBytes(max_workspace_size);
    hipblaslt_ext::Gemm gemm(
        handle, transa, transb, HIP_R_16BF, HIP_R_16BF, HIP_R_32F, HIP_R_32F, HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue
        epilogue; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a     = A;
    inputs.b     = B;
    inputs.c     = C;
    inputs.d     = D;
    inputs.alpha = &alpha;
    inputs.beta  = &beta;
    gemm.setProblem(m, n, k, batch_count, epilogue, inputs);

    const int                                     request_solutions = 1;
    std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
    CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));

    if(heuristicResult.empty())
    {
        std::cerr << "No valid solution found!" << std::endl;
        return;
    }

    // In this sample, the workspace is already allocated with max_workspace_size
    // If not, calculate the needed workspace_size and allocate d_workspace here
    // uint64_t workspace_size = 0;
    // for(int i = 0; i < returnedAlgoCount; i++)
    //     workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);
    // CHECK_HIP_ERRORhipMalloc(&d_workspace, workspace_size));

    // Make sure to initialize every time when algo changes
    CHECK_HIPBLASLT_ERROR(gemm.initialize(heuristicResult[0].algo, d_workspace));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));
    return;
}

// FP16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasLtHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int batch_count,
    float& alpha,
    float& beta,
    at::Half* A,
    at::Half* B,
    float*    C,
    float*    D,
    void*     d_workspace,
    int64_t   max_workspace_size,
    hipStream_t   stream) {
    hipblaslt_ext::GemmPreference gemmPref;
    gemmPref.setMaxWorkspaceBytes(max_workspace_size);
    hipblaslt_ext::Gemm gemm(
        handle, transa, transb, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F, HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue
        epilogue; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a     = A;
    inputs.b     = B;
    inputs.c     = C;
    inputs.d     = D;
    inputs.alpha = &alpha;
    inputs.beta  = &beta;
    gemm.setProblem(m, n, k, batch_count, epilogue, inputs);

    const int                                     request_solutions = 1;
    std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
    CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));

    if(heuristicResult.empty())
    {
        std::cerr << "No valid solution found!" << std::endl;
        return;
    }

    // In this sample, the workspace is already allocated with max_workspace_size
    // If not, calculate the needed workspace_size and allocate d_workspace here
    // uint64_t workspace_size = 0;
    // for(int i = 0; i < returnedAlgoCount; i++)
    //     workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);
    // CHECK_HIP_ERRORhipMalloc(&d_workspace, workspace_size));

    // Make sure to initialize every time when algo changes
    CHECK_HIPBLASLT_ERROR(gemm.initialize(heuristicResult[0].algo, d_workspace));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));
    return;
}


// FP32 wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasLtHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    int batch_count,
    float& alpha,
    float& beta,
    float* A,
    float* B,
    float* C,
    float* D,
    void*   d_workspace,
    int64_t  max_workspace_size,
    hipStream_t   stream) {
   hipblaslt_ext::GemmPreference gemmPref;
    gemmPref.setMaxWorkspaceBytes(max_workspace_size);
    hipblaslt_ext::Gemm gemm(
        handle, transa, transb, HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F, HIPBLAS_COMPUTE_32F);

    hipblaslt_ext::GemmEpilogue
        epilogue; // No action needed, default is HIPBLASLT_EPILOGUE_DEFAULT. (Gemm only)
    hipblaslt_ext::GemmInputs inputs;
    inputs.a     = A;
    inputs.b     = B;
    inputs.c     = C;
    inputs.d     = D;
    inputs.alpha = &alpha;
    inputs.beta  = &beta;
    gemm.setProblem(m, n, k, batch_count, epilogue, inputs);

    const int                                     request_solutions = 1;
    std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResult;
    CHECK_HIPBLASLT_ERROR(gemm.algoGetHeuristic(request_solutions, gemmPref, heuristicResult));

    if(heuristicResult.empty())
    {
        std::cerr << "No valid solution found!" << std::endl;
        return;
    }

    // In this sample, the workspace is already allocated with max_workspace_size
    // If not, calculate the needed workspace_size and allocate d_workspace here
    // uint64_t workspace_size = 0;
    // for(int i = 0; i < returnedAlgoCount; i++)
    //     workspace_size = max(workspace_size, heuristicResult[i].workspaceSize);
    // CHECK_HIP_ERRORhipMalloc(&d_workspace, workspace_size));

    // Make sure to initialize every time when algo changes
    CHECK_HIPBLASLT_ERROR(gemm.initialize(heuristicResult[0].algo, d_workspace));
    CHECK_HIPBLASLT_ERROR(gemm.run(stream));
    return;
}

template <typename T>
void wgrad_gemm_accum_fp32_cuda(T *input, T *d_output, float *dc_tensor, float *d_weight, int in_dim, int hidden_dim, int out_dim) {
    
    hipblasLtHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    float alpha = 1.0;
    float beta  = 1.0;
    const int batch_count = 1;
    void*   d_workspace;
    int64_t max_workspace_size = 32*1024*1024;
    if(max_workspace_size > 0)
        hipMalloc(&d_workspace, max_workspace_size);
    gemmex_wrapper(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,      //m
        out_dim,     // n
        hidden_dim,  //k
        batch_count,
        alpha,
        beta,
        input,      //da
        d_output,   //db
        dc_tensor, //dc
        d_weight,   //dd
        d_workspace,
        max_workspace_size,
        stream);
} 
    
template void wgrad_gemm_accum_fp32_cuda<at::Half>(at::Half *input, at::Half *d_output, float *dc_tensor, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, float *dc_tensor, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<float>(float *input, float *d_output, float *dc_tensor, float *d_weight, int in_dim, int hidden_dim, int out_dim);


void wgrad_gemm_accum_fp32_cuda_stub(
  at::Tensor &input,
  at::Tensor &d_output,
  at::Tensor &d_weight) 
{
    at::Tensor input_2d, d_output_2d;
    // input tensor: collapse to the first dim
    auto in_sizes = input.sizes();
    if (input.dim() > 2) {
        input_2d = input.view({-1, in_sizes[in_sizes.size() - 1]});
    } else {
        input_2d = input;
    }
    // d_output tensor: collapse to the first dim
    auto d_out_sizes = d_output.sizes();
    if (d_output.dim() > 2) {
        d_output_2d = d_output.view({-1, d_out_sizes[d_out_sizes.size() - 1]});
    } else {
        d_output_2d = d_output;
    }
    at::Tensor dc_tensor = at::empty_like(d_weight);
    dc_tensor.copy_(d_weight);
    //at::Tensor dc_tensor = at::zeros_like(d_weight);
    const int hidden_dim = input_2d.size(0);
    const int in_dim = input_2d.size(1);
    const int out_dim = d_weight.size(0);

    DISPATCH_FLOAT_HALF_AND_BFLOAT(input_2d.scalar_type(), 0, "wgrad_gemm_accum_fp32",
        wgrad_gemm_accum_fp32_cuda<scalar_t_0>(
            input_2d.data_ptr<scalar_t_0>(),
            d_output_2d.data_ptr<scalar_t_0>(),
            dc_tensor.data_ptr<float>(),
            d_weight.data_ptr<float>(),
            in_dim,
            hidden_dim,
            out_dim);
    );
}

/*
// BF16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// FP16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// FP32 wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta,
    float *C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

template <typename T>
void wgrad_gemm_accum_fp32_cuda(T *input, T *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha = 1.0;
    const float beta  = 1.0;

    gemmex_wrapper(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,
        out_dim,
        hidden_dim,
        &alpha,
        input,
        in_dim,
        d_output,
        out_dim,
        &beta,
        d_weight,
        in_dim);
}

template void wgrad_gemm_accum_fp32_cuda<at::Half>(at::Half *input, at::Half *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<float>(float *input, float *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);


void wgrad_gemm_accum_fp32_cuda_stub(
  at::Tensor &input,
  at::Tensor &d_output,
  at::Tensor &d_weight
) {
    at::Tensor input_2d, d_output_2d;
    // input tensor: collapse to the first dim
    auto in_sizes = input.sizes();
    if (input.dim() > 2) {
        input_2d = input.view({-1, in_sizes[in_sizes.size() - 1]});
    } else {
        input_2d = input;
    }
    // d_output tensor: collapse to the first dim
    auto d_out_sizes = d_output.sizes();
    if (d_output.dim() > 2) {
        d_output_2d = d_output.view({-1, d_out_sizes[d_out_sizes.size() - 1]});
    } else {
        d_output_2d = d_output;
    }

    const int hidden_dim = input_2d.size(0);
    const int in_dim = input_2d.size(1);
    const int out_dim = d_weight.size(0);

    DISPATCH_FLOAT_HALF_AND_BFLOAT(input_2d.scalar_type(), 0, "wgrad_gemm_accum_fp32",
        wgrad_gemm_accum_fp32_cuda<scalar_t_0>(
            input_2d.data_ptr<scalar_t_0>(),
            d_output_2d.data_ptr<scalar_t_0>(),
            d_weight.data_ptr<float>(),
            in_dim,
            hidden_dim,
            out_dim);
    );
}
*/